#include "hip/hip_runtime.h"
#include <imp/cu_core/cu_image_reduction.cuh>

namespace ze {
namespace cu {

// From Reduction SDK sample:
// Prevent instantiation of the generic struct using an undefined symbol
// in the function body (so it won't compile)
template<typename Pixel>
struct SharedMemory
{
  __device__ Pixel *getPointer()
  {
    extern __device__ void error(void);
    error();
    return NULL;
  }
};

// Required specializations
template<>
struct SharedMemory<Pixel32sC1>
{
  __device__ Pixel32sC1 *getPointer()
  {
    extern __shared__ Pixel32sC1 s_int[];
    return s_int;
  }
};

template<>
struct SharedMemory<Pixel32fC1>
{
  __device__ Pixel32fC1 *getPointer()
  {
    extern __shared__ Pixel32fC1 s_float[];
    return s_float;
  }
};

// Templated kernels
template<typename Pixel>
__global__
void reductionSumKernel(
    Pixel* out_dev_ptr,
    size_t out_stride,
    const Pixel* in_dev_ptr,
    size_t in_stride,
    uint32_t width,
    uint32_t height)
{
  SharedMemory<Pixel> smem;
  Pixel* s_partial = smem.getPointer();

  Pixel sum = 0;

  // Sum over 2D thread grid, use (x,y) indices
  for (int x = blockIdx.x * blockDim.x + threadIdx.x;
       x < width;
       x += blockDim.x * gridDim.x)
  {
    for (int y = blockIdx.y * blockDim.y + threadIdx.y;
         y < height;
         y += blockDim.y * gridDim.y)
    {
      sum += in_dev_ptr[y*in_stride+x];
    }
  }
  // Sums are written to shared memory, single index
  s_partial[threadIdx.y*blockDim.x+threadIdx.x] = sum;
  __syncthreads();

  // Reduce over block sums stored in shared memory
  // Start using half the block threads,
  // halve the active threads at each iteration
  const int tid = threadIdx.y*blockDim.x+threadIdx.x;
  for (int num_active_threads = (blockDim.x*blockDim.y) >> 1;
       num_active_threads;
       num_active_threads >>= 1)
  {
    if (tid < num_active_threads)
    {
      s_partial[tid] += s_partial[tid+num_active_threads];
    }
    __syncthreads();
  }
  // Thread 0 writes the result for the block
  if (0 == tid)
  {
    out_dev_ptr[blockIdx.y*out_stride+blockIdx.x] = s_partial[0];
  }
}


template<typename Pixel>
__global__
void reductionCountEqKernel(
    Pixel* out_dev_ptr,
    size_t out_stride,
    const Pixel* in_dev_ptr,
    size_t in_stride,
    uint32_t width,
    uint32_t height,
    Pixel value)
{
  SharedMemory<Pixel> smem;
  Pixel* s_partial = smem.getPointer();

  int32_t count = 0;

  // Sum over 2D thread grid, use (x,y) indices
  for (int x = blockIdx.x * blockDim.x + threadIdx.x;
       x < width;
       x += blockDim.x * gridDim.x)
  {
    for (int y = blockIdx.y * blockDim.y + threadIdx.y;
         y < height;
         y += blockDim.y * gridDim.y)
    {
      if(static_cast<int32_t>(value) == in_dev_ptr[y*in_stride+x])
      {
        count += 1;
      }
    }
  }
  // Sums are written to shared memory, single index
  s_partial[threadIdx.y*blockDim.x+threadIdx.x] = count;
  __syncthreads();

  // Reduce over block sums stored in shared memory
  // Start using half the block threads,
  // halve the active threads at each iteration
  const int tid = threadIdx.y*blockDim.x+threadIdx.x;
  for (int num_active_threads = (blockDim.x*blockDim.y) >> 1;
       num_active_threads;
       num_active_threads >>= 1 )
  {
    if (tid < num_active_threads)
    {
      s_partial[tid] += s_partial[tid+num_active_threads];
    }
    __syncthreads();
  }
  // Thread 0 writes the result for the block
  if (0 == tid)
  {
    out_dev_ptr[blockIdx.y*out_stride+blockIdx.x] = s_partial[0];
  }
}


template<typename Pixel>
ImageReducer<Pixel>::ImageReducer()
  : partial_(fragm_.dimGrid.x, fragm_.dimGrid.y)
{
  // Compute required amount of shared memory
  sh_mem_size_ = fragm_.dimBlock.x * fragm_.dimBlock.y * sizeof(Pixel);
}

template<typename Pixel>
ImageReducer<Pixel>::~ImageReducer()
{ }

// Sum image by reduction
// Cfr. listing 12.1 by N. Wilt, "The CUDA Handbook"
template<typename Pixel>
Pixel ImageReducer<Pixel>::sum(const ImageGpu<Pixel>& in_img)
{
  //if(is_dev_fin_alloc_ && is_dev_part_alloc_)

  reductionSumKernel<Pixel>
      <<<
        fragm_.dimGrid, fragm_.dimBlock, sh_mem_size_
      >>>
        (partial_.data(),
         partial_.stride(),
         in_img.data(),
         in_img.stride(),
         in_img.width(),
         in_img.height());

  reductionSumKernel<Pixel>
      <<<
        1, fragm_.dimBlock, sh_mem_size_
      >>>
        (dev_final_.data(),
         0,
         partial_.data(),
         partial_.stride(),
         fragm_.dimGrid.x,
         fragm_.dimGrid.y);

  // download sum
  ze::LinearMemory<Pixel> h_sum(1);
  dev_final_.copyTo(h_sum);
  return h_sum(0);
}

// Count elements equal to 'value'
// First count over the thread grid,
// then perform a reduction sum on a single thread block
template<>
size_t ImageReducer<Pixel32sC1>::countEqual(
    const ImageGpu32sC1& in_img,
    int32_t value)
{

  reductionCountEqKernel<Pixel32sC1>
      <<<
        fragm_.dimGrid, fragm_.dimBlock, sh_mem_size_
      >>>
        (partial_.data(),
         partial_.stride(),
         in_img.data(),
         in_img.stride(),
         in_img.width(),
         in_img.height(),
         value);

  reductionSumKernel<Pixel32sC1>
      <<<
        1, fragm_.dimBlock, sh_mem_size_
      >>>
        (dev_final_.data(),
         0,
         partial_.data(),
         partial_.stride(),
         fragm_.dimGrid.x,
         fragm_.dimGrid.y);

  // download count
  ze::LinearMemory32sC1 h_count{1};
  dev_final_.copyTo(h_count);
  return static_cast<size_t>(h_count(0));
}

template class ImageReducer<Pixel32sC1>;
template class ImageReducer<Pixel32fC1>;

} // cu namespace
} // ze namespace
