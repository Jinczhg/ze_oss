#include <imp/cu_core/cu_linearmemory.cuh>

#include <imp/cu_core/cu_exception.hpp>
#include <imp/cu_core/cu_utils.hpp>
#include <imp/cu_core/cu_k_setvalue.cuh>


namespace ze {
namespace cu {

//-----------------------------------------------------------------------------
template<typename Pixel>
LinearMemory<Pixel>::LinearMemory(const std::uint32_t& length)
  : LinearMemoryBase(length)
  , data_(Memory::alloc(this->length()))
{
}

//-----------------------------------------------------------------------------
template<typename Pixel>
LinearMemory<Pixel>::LinearMemory(const ze::cu::LinearMemory<Pixel>& from)
  : ze::cu::LinearMemory<Pixel>(from.length())
{
  if (from.data() == 0)
  {
    throw ze::cu::Exception("'from' data not valid", __FILE__, __FUNCTION__, __LINE__);
  }
  this->copyFrom(from);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
LinearMemory<Pixel>::LinearMemory(const ze::LinearMemory<Pixel>& from)
  : ze::cu::LinearMemory<Pixel>(from.length())
{
  if (from.data() == 0)
  {
    throw ze::cu::Exception("'from' data not valid", __FILE__, __FUNCTION__, __LINE__);
  }
  this->copyFrom(from);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
Pixel* LinearMemory<Pixel>::data()
{
  return data_.get();
}

//-----------------------------------------------------------------------------
template<typename Pixel>
const Pixel* LinearMemory<Pixel>::data() const
{
  return reinterpret_cast<const Pixel*>(data_.get());
}

//-----------------------------------------------------------------------------
template<typename Pixel>
auto LinearMemory<Pixel>::cuData() -> decltype(ze::cu::toCudaVectorType(this->data()))
{
  return ze::cu::toCudaVectorType(this->data());
}

//-----------------------------------------------------------------------------
template<typename Pixel>
auto LinearMemory<Pixel>::cuData() const -> decltype(ze::cu::toConstCudaVectorType(this->data()))
{
  return ze::cu::toConstCudaVectorType(this->data());
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::setValue(const Pixel& value)
{
  if (sizeof(Pixel) == 1)
  {
    hipMemset((void*)(this->data()+this->roi().x()), (int)value.c[0], this->roiBytes());
  }
  else
  {
    // fragmentation
    cu::Fragmentation<32,1> frag(this->roi().length());

    // todo add roi to kernel!
    ze::cu::k_setValue
        <<< frag.dimGrid, frag.dimBlock
        >>> (this->data(), this->roi().x(), this->roi().length(), value);
  }
  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyTo(ze::cu::LinearMemory<Pixel>& dst)
{
  if (dst.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->roiBytes() != dst.roiBytes())
    IMP_THROW_EXCEPTION("source and destination array region of interests are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(dst.data()+dst.roi().x(), this->data()+this->roi().x(),
                 this->roiBytes(), hipMemcpyDeviceToDevice);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyFrom(const ze::cu::LinearMemory<Pixel>& from)
{
  if (from.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->roiBytes() != from.roiBytes())
    IMP_THROW_EXCEPTION("source and destination array region of interests are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(this->data()+this->roi().x(), from.data()+from.roi().x(),
                 from.roiBytes(), hipMemcpyDeviceToDevice);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyTo(ze::LinearMemory<Pixel>& dst)
{
  if (dst.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->roiBytes() != dst.roiBytes())
    IMP_THROW_EXCEPTION("source and destination array region of interests are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(dst.data()+dst.roi().x(), this->data()+this->roi().x(),
                 this->roiBytes(), hipMemcpyDeviceToHost);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyFrom(const ze::LinearMemory<Pixel>& from)
{
  if (from.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->roiBytes() != from.roiBytes())
    IMP_THROW_EXCEPTION("source and destination array region of interests are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(this->data()+this->roi().x(), from.data()+from.roi().x(),
                 from.roiBytes(), hipMemcpyHostToDevice);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}


//=============================================================================
// Explicitely instantiate the desired classes
template class LinearMemory<ze::Pixel8uC1>;
template class LinearMemory<ze::Pixel8uC2>;
template class LinearMemory<ze::Pixel8uC3>;
template class LinearMemory<ze::Pixel8uC4>;

template class LinearMemory<ze::Pixel16uC1>;
template class LinearMemory<ze::Pixel16uC2>;
template class LinearMemory<ze::Pixel16uC3>;
template class LinearMemory<ze::Pixel16uC4>;

template class LinearMemory<ze::Pixel32uC1>;
template class LinearMemory<ze::Pixel32uC2>;
template class LinearMemory<ze::Pixel32uC3>;
template class LinearMemory<ze::Pixel32uC4>;

template class LinearMemory<ze::Pixel32sC1>;
template class LinearMemory<ze::Pixel32sC2>;
template class LinearMemory<ze::Pixel32sC3>;
template class LinearMemory<ze::Pixel32sC4>;

template class LinearMemory<ze::Pixel32fC1>;
template class LinearMemory<ze::Pixel32fC2>;
template class LinearMemory<ze::Pixel32fC3>;
template class LinearMemory<ze::Pixel32fC4>;

} // namespace cu
} // namespace ze
