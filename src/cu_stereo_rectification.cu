#include "hip/hip_runtime.h"
#include <imp/cu_core/cu_linearmemory.cuh>
#include <imp/cu_core/cu_texture.cuh>
#include <imp/cu_imgproc/cu_remap.cuh>
#include <imp/cu_imgproc/cu_stereo_rectification.cuh>
#include <ze/geometry/epipolar_geometry.hpp>

namespace ze {
namespace cu {

//! @todo (MPI) test constant memory fo camera/dist parameters
//! maybe also for the rectifying homography
template<typename CameraModel,
         typename DistortionModel>
__global__
void k_computeUndistortRectifyMap(
    Pixel32fC2* dst,
    size_t dst_stride,
    std::uint32_t width,
    std::uint32_t height,
    const float* d_cam_params,
    const float* d_transformed_cam_params,
    const float* d_dist_coeffs,
    const float* d_inv_H)
{
  const int u = blockIdx.x*blockDim.x + threadIdx.x;
  const int v = blockIdx.y*blockDim.y + threadIdx.y;

  if (u < width && v < height)
  {
    float px[2]{static_cast<float>(u), static_cast<float>(v)};
    CameraModel::backProject(d_transformed_cam_params, px);
    const float x = d_inv_H[0]*px[0]+d_inv_H[3]*px[1]+d_inv_H[6];
    const float y = d_inv_H[1]*px[0]+d_inv_H[4]*px[1]+d_inv_H[7];
    const float w = d_inv_H[2]*px[0]+d_inv_H[5]*px[1]+d_inv_H[8];
    px[0] = x / w;
    px[1] = y / w;
    DistortionModel::distort(d_dist_coeffs, px);
    CameraModel::project(d_cam_params, px);
    dst[v*dst_stride + u][0] = px[0];
    dst[v*dst_stride + u][1] = px[1];
  }
}

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
StereoRectifier<CameraModel, DistortionModel, Pixel>::StereoRectifier(
    Size2u img_size,
    Vector4& camera_params,
    Vector4& transformed_camera_params,
    Vector4& dist_coeffs,
    Matrix3& inv_H)
  : undistort_rectify_map_(img_size)
  , fragm_(img_size)
{
  //! Upload to GPU
  //! Convert to float
  Eigen::Vector4f cp_flt = camera_params.cast<float>();
  Eigen::Vector4f tcp_flt = transformed_camera_params.cast<float>();
  Eigen::Vector4f dist_flt = dist_coeffs.cast<float>();
  Eigen::Matrix3f inv_H_flt = inv_H.cast<float>();
  //! Copy to host LinearMemory
  ze::LinearMemory32fC1 h_cam_params(
        reinterpret_cast<Pixel32fC1*>(cp_flt.data()),
        4, true);
  ze::LinearMemory32fC1 h_transformed_cam_params(
        reinterpret_cast<Pixel32fC1*>(tcp_flt.data()),
        4, true);
  ze::LinearMemory32fC1 h_dist_coeffs(
        reinterpret_cast<Pixel32fC1*>(dist_flt.data()),
        4, true);
  ze::LinearMemory32fC1 h_inv_H(
        reinterpret_cast<Pixel32fC1*>(inv_H_flt.data()),
        9, true);
  //! Copy to device LinearMemory
  cu::LinearMemory32fC1 d_cam_params(h_cam_params);
  cu::LinearMemory32fC1 d_transformed_cam_params(h_transformed_cam_params);
  cu::LinearMemory32fC1 d_dist_coeffs(h_dist_coeffs);
  cu::LinearMemory32fC1 d_inv_H(h_inv_H);

  //! Compute map
  k_computeUndistortRectifyMap<CameraModel, DistortionModel>
      <<<
        fragm_.dimGrid, fragm_.dimBlock
      >>> (undistort_rectify_map_.data(),
           undistort_rectify_map_.stride(),
           undistort_rectify_map_.width(),
           undistort_rectify_map_.height(),
           d_cam_params.cuData(),
           d_transformed_cam_params.cuData(),
           d_dist_coeffs.cuData(),
           d_inv_H.cuData());
}

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
void StereoRectifier<CameraModel, DistortionModel, Pixel>::rectify(
    ImageGpu<Pixel>& dst,
    const ImageGpu<Pixel>& src) const
{
  CHECK_EQ(src.size(), dst.size());
  CHECK_EQ(src.size(), undistort_rectify_map_.size());

  // Attach texture
  std::shared_ptr<Texture2D> src_tex =
      src.genTexture(false, hipFilterModeLinear);
  IMP_CUDA_CHECK();

  //! Execute remapping
  k_remap
      <<<
        fragm_.dimGrid, fragm_.dimBlock
      >>> (dst.data(),
           dst.stride(),
           undistort_rectify_map_.data(),
           undistort_rectify_map_.stride(),
           dst.width(),
           dst.height(),
           *src_tex);
  IMP_CUDA_CHECK();
}

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
const ImageGpu32fC2& StereoRectifier<CameraModel, DistortionModel, Pixel>::getUndistortRectifyMap() const
{
  return undistort_rectify_map_;
}

// Explicit template instantiations
template class StereoRectifier<PinholeGeometry, EquidistantDistortion, Pixel32fC1>;
template class StereoRectifier<PinholeGeometry, RadialTangentialDistortion, Pixel32fC1>;

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
HorizontalStereoPairRectifier<CameraModel, DistortionModel, Pixel>::HorizontalStereoPairRectifier(
    Size2u img_size,
    Vector4& left_camera_params,
    Vector4& transformed_left_cam_params,
    Vector4& left_dist_coeffs,
    Vector4& right_camera_params,
    Vector4& transformed_right_cam_params,
    Vector4& right_dist_coeffs,
    Transformation& T_L_R,
    FloatType& horizontal_offset)
{
  Matrix3 left_H;
  Matrix3 right_H;

  computeHorizontalStereoParameters
      <CameraModel, DistortionModel>(
        img_size,
        left_camera_params,
        left_dist_coeffs,
        right_camera_params,
        right_dist_coeffs,
        T_L_R,
        left_H,
        right_H,
        transformed_left_cam_params,
        transformed_right_cam_params,
        horizontal_offset);

  //! Allocate rectifiers for the left and right cameras
  Matrix3 inv_left_H = left_H.inverse();
  Matrix3 inv_right_H = right_H.inverse();
  left_rectifier_.reset(
        new StereoRectifier<CameraModel, DistortionModel, Pixel>(
          img_size, left_camera_params, transformed_left_cam_params, left_dist_coeffs, inv_left_H));
  right_rectifier_.reset(
        new StereoRectifier<CameraModel, DistortionModel, Pixel>(
          img_size, right_camera_params, transformed_right_cam_params, right_dist_coeffs, inv_right_H));
}

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
void HorizontalStereoPairRectifier<CameraModel, DistortionModel, Pixel>::rectify(
    ImageGpu<Pixel>& left_dst,
    ImageGpu<Pixel>& right_dst,
    const ImageGpu<Pixel>& left_src,
    const ImageGpu<Pixel>& right_src) const
{
  left_rectifier_->rectify(left_dst, left_src);
  right_rectifier_->rectify(right_dst, right_src);
}

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
const ImageGpu32fC2& HorizontalStereoPairRectifier<CameraModel, DistortionModel, Pixel>::getLeftCameraUndistortRectifyMap() const
{
  return left_rectifier_->getUndistortRectifyMap();
}

template <typename CameraModel,
          typename DistortionModel,
          typename Pixel>
const ImageGpu32fC2& HorizontalStereoPairRectifier<CameraModel, DistortionModel, Pixel>::getRightCameraUndistortRectifyMap() const
{
  return right_rectifier_->getUndistortRectifyMap();
}

// Explicit template instantiations
template class HorizontalStereoPairRectifier<PinholeGeometry, EquidistantDistortion, Pixel32fC1>;
template class HorizontalStereoPairRectifier<PinholeGeometry, RadialTangentialDistortion, Pixel32fC1>;

} // cu namespace
} // ze namespace
